#include "hip/hip_runtime.h"
/*
author: cltian
date: 2018/4/11
refs:
     http://answers.opencv.org/question/89050/passing-an-array-of-cvgpumat-to-a-cuda-kernel/
     http://answers.opencv.org/question/8466/how-access-gpumat-in-a-kernel/
     http://answers.opencv.org/question/26059/using-gpu-module-with-own-code/
     https://stackoverflow.com/questions/6965465/how-to-convert-gpumat-to-cvmat-in-opencv
     https://hk.saowen.com/a/43287d4469fb06d8e398c2b78572e8165133d687cac9082916ee12c69b44131d
*/

#include "test.h"
#include "hip/hip_vector_types.h"

using namespace cv;
using namespace cv::gpu;

__global__ void solveAllKernel(PtrStepSz<double>* pdSrc, double* sWgt, int cA1, int cA2, int cA3, int lvl) {
                
  int d = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int x = threadIdx.z + blockDim.z * blockIdx.z;

  if (d<cA1 && y<cA2 && x<cA3){
      int curY = y;
      int curX = x;
      int curD = d;
      double sum = 0;
      
      for( int s = 0; s < lvl; s ++ ) {
        //int idx = curY*imgW+curX;
        double curCost = pdSrc[s*cA1+curD].ptr(curY)[curX];
        //double curCost = smPyr[ s ]->costVol[ curD ].at<double>( curY, curX );
        sum += sWgt[ s ] * curCost;
        curY = curY / 2;
        curX = curX / 2;
        curD = ( curD + 1 ) / 2;
      }
      //int idxAss = assY*imgW+assX;
      pdSrc[m*cA1+d].ptr(y)[x] = sum;
      //newCosts[ m ][ assD ].at<double>( assY, assX ) = sum;
  }
}

