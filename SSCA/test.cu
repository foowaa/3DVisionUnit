#include "hip/hip_runtime.h"
/*
author: cltian
date: 2018/4/11
refs:
     http://answers.opencv.org/question/89050/passing-an-array-of-cvgpumat-to-a-cuda-kernel/
     http://answers.opencv.org/question/8466/how-access-gpumat-in-a-kernel/
     http://answers.opencv.org/question/26059/using-gpu-module-with-own-code/
     https://stackoverflow.com/questions/6965465/how-to-convert-gpumat-to-cvmat-in-opencv
     https://hk.saowen.com/a/43287d4469fb06d8e398c2b78572e8165133d687cac9082916ee12c69b44131d
*/

#include "test.h"
#include "hip/hip_vector_types.h"

using namespace cv;
using namespace cv::gpu;

__global__ void solveAllKernel(PtrStepSz<double>* pdSrc, double* sWgt, int cA1, int cA2, int cA3, int lvl) {
                
  int d = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int x = threadIdx.z + blockDim.z * blockIdx.z;

  if (d<cA1 && y<cA2 && x<cA3){
      int curY = y;
      int curX = x;
      int curD = d;
      double sum = 0;
      
      for( int s = 0; s < lvl; s ++ ) {
        //int idx = curY*imgW+curX;
        double curCost = pdSrc[s*cA1+curD].ptr(curY)[curX];
        //double curCost = smPyr[ s ]->costVol[ curD ].at<double>( curY, curX );
        sum += sWgt[ s ] * curCost;
        curY = curY / 2;
        curX = curX / 2;
        curD = ( curD + 1 ) / 2;
      }
      //int idxAss = assY*imgW+assX;
      pdSrc[m*cA1+d].ptr(y)[x] = sum;
      //newCosts[ m ][ assD ].at<double>( assY, assX ) = sum;
  }
}


void caller_test(SSCA**& smPyr, double* invWgt, int lvl){
    	GpuMat* mats;
	const int constVar1 = PY_LVL*smPyr[ 0 ]->maxDis;
	const int constVar2 = smPyr[ 0 ]->maxDis;
	PtrStepSz<double>* phSrc = new PtrStepSz<double>[constVar1];
	PtrStepSz<double>* phDst = new PtrStepSz<double>[constVar1];
	PtrStepSz<double>* pdSrc;
	//PtrStepSz<double>* pdDst;
	double *dinvWgt  = new double[ PY_LVL * PY_LVL];
	int i = 0;
	for(int s=0; s<PY_LVL; s++)
		for(int d=0; d<constVar2; d++)
			mats[i++].upload(smPyr[s]->rcostVol[d]);
	for (int i=0; i<constVar1; i++)
		phSrc[i] = mats[i];
    cudaCheckError(hipMalloc(&dinvWgt, sizeof(double)*PY_LVL*PY_LVL));
	cudaCheckError(hipMalloc(&pdSrc, constVar1*sizeof(PtrStepSz<double>)));
	//cudaCheckError(hipMalloc(&pdDst, constVar1*sizeof(PtrStepSz<double>)));
    cudaCheckError(hipMemcpy(dinvWgt, invWgt, sizeof(double)*PY_LVL*PY_LVL, hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(pdSrc, phSrc, constVar1*sizeof(PtrStepSz<double>), 
		                      hipMemcpyHostToDevice));

    dim3 block(8, 8, 8);
    dim3 grid( (constVar2+block.x-1)/block.x, (hei+block.y-1)/block.y, (wid+block.z-1)/block.z);

	solveAllKernel<<<grid, block>>>(pdSrc, dinvWgt, constVar2, hei, wid, PY_LVL);

	cudaCheckError(hipMemcpy(phDst, pdSrc, constVar1*sizeof(PtrStepSz<double>),
		                      hipMemcpyDeviceToHost));
	for (int i=0; i<constVar1; i++)
		mats[i] = phDst[i];
	for( int s = 0; s < PY_LVL; s ++ ) {
		for( int d = 0; d < smPyr[ s ]->maxDis; d ++ ) {
			//smPyr[ s ]->costVol[ d ] = pDistData[ s*PY_LVL+d ]
			Mat tempMatrix;
		        mats[s*PY_LVL+d].download(tempMatrix);
		    smPyr[ s ]->costVol[ d ] = tempMatrix.clone();
		}
	}

	cudaCheckError(hipFree(pdSrc));
	cudaCheckError(hipFree(dinvWgt));
}


void caller_test(SSCA**& smPyr, double* invWgt, int lvl){
    	GpuMat* mats;
	const int constVar1 = PY_LVL*smPyr[ 0 ]->maxDis;
	const int constVar2 = smPyr[ 0 ]->maxDis;
	PtrStepSz<double>* phSrc = new PtrStepSz<double>[constVar1];
	PtrStepSz<double>* phDst = new PtrStepSz<double>[constVar1];
	PtrStepSz<double>* pdSrc;
	//PtrStepSz<double>* pdDst;
	double *dinvWgt  = new double[ PY_LVL * PY_LVL];
	int i = 0;
	for(int s=0; s<PY_LVL; s++)
		for(int d=0; d<constVar2; d++)
			mats[i++].upload(smPyr[s]->costVol[d]);
	for (int i=0; i<constVar1; i++)
		phSrc[i] = mats[i];
    cudaCheckError(hipMalloc(&dinvWgt, sizeof(double)*PY_LVL*PY_LVL));
	cudaCheckError(hipMalloc(&pdSrc, constVar1*sizeof(PtrStepSz<double>)));
	//cudaCheckError(hipMalloc(&pdDst, constVar1*sizeof(PtrStepSz<double>)));
    cudaCheckError(hipMemcpy(dinvWgt, invWgt, sizeof(double)*PY_LVL*PY_LVL, hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(pdSrc, phSrc, constVar1*sizeof(PtrStepSz<double>), 
		                      hipMemcpyHostToDevice));

    dim3 block(8, 8, 8);
    dim3 grid( (constVar2+block.x-1)/block.x, (hei+block.y-1)/block.y, (wid+block.z-1)/block.z);

	solveAllKernel<<<grid, block>>>(pdSrc, dinvWgt, constVar2, hei, wid, PY_LVL);

	cudaCheckError(hipMemcpy(phDst, pdSrc, constVar1*sizeof(PtrStepSz<double>),
		                      hipMemcpyDeviceToHost));
	for (int i=0; i<constVar1; i++)
		mats[i] = phDst[i];
	for( int s = 0; s < PY_LVL; s ++ ) {
		for( int d = 0; d < smPyr[ s ]->maxDis; d ++ ) {
			//smPyr[ s ]->costVol[ d ] = pDistData[ s*PY_LVL+d ]
			Mat tempMatrix;
		        mats[s*PY_LVL+d].download(tempMatrix);
		    smPyr[ s ]->costVol[ d ] = tempMatrix.clone();
		}
	}

	cudaCheckError(hipFree(pdSrc));
	cudaCheckError(hipFree(dinvWgt));
}

