#include "hip/hip_runtime.h"
/*
author: cltian
date: 2018/4/11
refs:
     http://answers.opencv.org/question/89050/passing-an-array-of-cvgpumat-to-a-cuda-kernel/
     http://answers.opencv.org/question/8466/how-access-gpumat-in-a-kernel/
     http://answers.opencv.org/question/26059/using-gpu-module-with-own-code/
     https://stackoverflow.com/questions/6965465/how-to-convert-gpumat-to-cvmat-in-opencv
     https://hk.saowen.com/a/43287d4469fb06d8e398c2b78572e8165133d687cac9082916ee12c69b44131d
*/

#include "test.h"
#include "hip/hip_vector_types.h"



using namespace cv;
using namespace cv::gpu;

__global__ void solveAllKernel(PtrStepSz<double>* pdSrc, PtrStepSz<double>* pdDst, double* invWgt, 
                               int imgW, int imgH, int cA1, int cA2, int cA3, int lvl) {
                
  int d = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int x = threadIdx.z + blockDim.z * blockIdx.z;

  if (d<cA1 && y<cA2 && x<cA3){
    for( int m = 0; m < lvl; m ++ ) {
      double sum = 0.0f;
      double* sWgt = invWgt + m * lvl;
      int curY = y;
      int curX = x;
      int curD = d;
      int assY = y;
      int assX = x;
      int assD = d;
      for( int s = 0; s < lvl; s ++ ) {
        if( s == m ) {
          assY = curY;
          assX = curX;
          assD = curD;
        }
        //int idx = curY*imgW+curX;
        double curCost = pdSrc[s*cA1+curD].ptr(curY)[curX];
        //double curCost = smPyr[ s ]->costVol[ curD ].at<double>( curY, curX );
        sum += sWgt[ s ] * curCost;
        curY = curY / 2;
        curX = curX / 2;
        curD = ( curD + 1 ) / 2;
      }
      //int idxAss = assY*imgW+assX;
      pdDst[m*cA1+assD].ptr(assY)[assX] = sum;
      //newCosts[ m ][ assD ].at<double>( assY, assX ) = sum;
    }
  }
}

